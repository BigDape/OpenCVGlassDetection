#include "hip/hip_runtime.h"
#include "kernel.h"

// CUDA 核函数，处理图像分类
__global__ void processClassificationKernel(cv::cuda::GpuMat* gpuImages,
                                            int tcount,
                                            cv::dnn::Net* net,
                                            int* type,
                                            float* pixLength,
                                            float* pixWidth,
                                            float* pixX,
                                            float* pixY)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < tcount) {
        // cv::cuda::GpuMat image = gpuImages[tid];
        // cv::cuda::GpuMat threeChannelImage;
        // cv::cuda::cvtColor(image, threeChannelImage, cv::COLOR_BGRA2BGR);
//         if (threeChannelImage.empty()) {
//             return;
//         }

        // cv::cuda::GpuMat blob;
        // cv::dnn::blobFromImage(threeChannelImage, blob, 1.0 / 255.0, cv::Size(640, 640), cv::Scalar(), true, false);

//         // 准备输入
        // std::vector<cv::cuda::GpuMat> image1;
//         net->setInput(gpuImages[tid]);

// //         // 前向传播并获取输出
//         std::vector<cv::cuda::GpuMat> outputs;
//         net->forward(outputs, net->getUnconnectedOutLayersNames());

//         int rows = outputs[0].size[1];
//         int dimensions = outputs[0].size[2];
//         if (dimensions > rows) {
//             rows = outputs[0].size[2];
//             dimensions = outputs[0].size[1];
//             outputs[0] = outputs[0].reshape(1, dimensions);
//             cv::transpose(outputs[0], outputs[0]);
//         }

//         float* data = (float*)outputs[0].data;
        // std::vector<int> class_ids;
        // std::vector<float> confidences;
        // std::vector<cv::Rect> boxes;

        // std::vector<std::string> classes = { "JieShi_Min", "MaoXu_Min", "QiPao_Min", "ShuiDi_Min", "BoLiXue_Min", "HeiDian_Min", "HuaShang_Min" };
        // double modelScoreThreshold = 0.1;
        // double modelConfidenceThreshold = 0.1;

        // cv::Point class_id1;
        // double maxClassScore1;
        // cv::Mat res1 = outputs[0](cv::Rect(4, 0, 7, rows));//7==classes.size()
        // cv::cuda::minMaxLoc(res1, 0, &maxClassScore1, 0, &class_id1);
        // data += dimensions * class_id1.y;
        // type[tid] = class_id1.x;
        // pixX[tid] = data[0];
        // pixY[tid] = data[1];
        // pixWidth[tid] = data[2];
        // pixLength[tid] = data[3];
    }
}

void cudaProcessClassification(cv::Mat defectimage, cv::dnn::Net net)
{
//     NewDefectUnitData resdata;
//     cv::Mat* defectimage_dev;
//     cv::dnn::Net* net_dev;
//     NewDefectUnitData* resdata_dev;

//     // 将输入图像、网络和输出数据复制到 GPU 内存
//     hipMalloc(&defectimage_dev, sizeof(cv::Mat));
//     hipMemcpy(defectimage_dev, &defectimage, sizeof(cv::Mat), hipMemcpyHostToDevice);
//     hipMalloc(&net_dev, sizeof(cv::dnn::Net));
//     hipMemcpy(net_dev, &net, sizeof(cv::dnn::Net), hipMemcpyHostToDevice);
//     hipMalloc(&resdata_dev, sizeof(NewDefectUnitData));

//     // 调用 CUDA 核函数
//     dim3 blockSize(1);
//     dim3 gridSize(1);
//     processClassificationKernel<<<gridSize, blockSize>>>(defectimage_dev, net_dev, resdata_dev);

//     // 将结果从 GPU 内存复制回主机内存
//     hipMemcpy(&resdata, resdata_dev, sizeof(NewDefectUnitData), hipMemcpyDeviceToHost);

//     // 释放 GPU 内存
//     hipFree(defectimage_dev);
//     hipFree(net_dev);
//     hipFree(resdata_dev);
}
